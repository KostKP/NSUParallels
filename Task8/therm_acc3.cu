#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <cstring>
#include <chrono>
#include <fstream>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <cub/block/block_reduce.cuh>

using namespace std;
using namespace chrono;

#define OUT_FILE "result.dat"

#ifndef NX
#define NX 30
#endif
#ifndef NY
#define NY 20
#endif

#define TAU -0.01
#define EPS 0.01
#define MAX_ITER 1000
#define SIZE ((size_t)(NX) * (size_t)(NY))
#define BLOCK_SIZE 256

__device__ double get_a(int row, int col) {
    if (row == col) return -4;
    if (row + 1 == col) return 1;
    if (row - 1 == col) return 1;
    if (row + NX == col) return 1;
    if (row - NX == col) return 1;
    return 0;
}

__device__ double get_b(int idx) {
    if (idx == NY / 2 * NX + NX / 3) return 10;
    if (idx == NY * 2 / 3 * NX + NX * 2 / 3) return -25;
    return 0;
}

__global__ void init_b(double* b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < SIZE) {
        b[idx] = get_b(idx);
    }
}

__global__ void mul_mv_sub(double* res, const double* x, const double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= SIZE) return;

    double tmp = -b[i];
    for (int j = 0; j < SIZE; ++j) {
        double a_ij = get_a(i, j);
        if (a_ij != 0.0) {
            tmp += a_ij * x[j];
        }
    }
    res[i] = tmp;
}

__global__ void update_x(double* x, const double* delta) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SIZE) {
        x[i] -= TAU * delta[i];
    }
}

__global__ void block_reduce_norm(const double* vec, double* block_results) {
    __shared__ typename hipcub::BlockReduce<double, BLOCK_SIZE>::TempStorage temp_storage;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double val = (i < SIZE) ? vec[i] * vec[i] : 0.0;

    double sum = hipcub::BlockReduce<double, BLOCK_SIZE>(temp_storage).Sum(val);

    if (threadIdx.x == 0) block_results[blockIdx.x] = sum;
}

double compute_norm(double* d_vec, double* d_temp, int size) {
    int blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    block_reduce_norm<<<blocks, BLOCK_SIZE>>>(d_vec, d_temp);
    double* h_temp = new double[blocks];
    hipMemcpy(h_temp, d_temp, blocks * sizeof(double), hipMemcpyDeviceToHost);
    double total = 0.0;
    for (int i = 0; i < blocks; i++) total += h_temp[i];
    delete[] h_temp;
    return sqrt(total);
}

int main() {
    double *d_x, *d_b, *d_Axmb, *d_temp;
    hipMalloc(&d_x, SIZE * sizeof(double));
    hipMalloc(&d_b, SIZE * sizeof(double));
    hipMalloc(&d_Axmb, SIZE * sizeof(double));
    hipMalloc(&d_temp, ((SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE) * sizeof(double));

    hipMemset(d_x, 0, SIZE * sizeof(double));
    init_b<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_b);

    int iterations = 0;
    double final_norm = 0.0;

    double norm_b = compute_norm(d_b, d_temp, SIZE);

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipGraph_t graph;
    hipGraphExec_t instance;

    bool graph_created = false;

    auto start = high_resolution_clock::now();

    while (iterations < MAX_ITER) {
        if (!graph_created) {
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

            mul_mv_sub<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(d_Axmb, d_x, d_b);
            update_x<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(d_x, d_Axmb);

            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
            graph_created = true;
        }

        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);

        mul_mv_sub<<<(SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_Axmb, d_x, d_b);
        final_norm = compute_norm(d_Axmb, d_temp, SIZE);

        double rel_norm = final_norm / norm_b;
        printf("%lf >= %lf\r", rel_norm, EPS);
        fflush(stdout);
        if (rel_norm < EPS) break;

        iterations++;
    }
    
    printf("\33[2K\r");
    fflush(stdout);

    auto end = high_resolution_clock::now();
    duration<double> diff = end - start;

    double* h_x = new double[SIZE];
    hipMemcpy(h_x, d_x, SIZE * sizeof(double), hipMemcpyDeviceToHost);

    printf("Matrix %dx%d processing time: %.4f sec. (Iterations: %d/%d, Final norm: %.8lf < %.8lf).\n",
           NX, NY, diff.count(), iterations, MAX_ITER, final_norm / norm_b, EPS);

    if ((NX == 10 && NY == 10) || (NX == 13 && NY == 13)) {
        cout << "Result matrix:\n";
        for (int i = 0; i < NY; ++i) {
            for (int j = 0; j < NX; ++j) {
                cout << fixed << setw(10) << setprecision(4) << h_x[i * NX + j] << " ";
            }
            cout << endl;
        }
    }

    FILE* f = fopen(OUT_FILE, "wb");
    fwrite(h_x, sizeof(double), SIZE, f);
    fclose(f);
    printf("Result matrix saved to file '%s'.\n", OUT_FILE);

    delete[] h_x;

    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_Axmb);
    hipFree(d_temp);
    hipGraphExecDestroy(instance);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);

    return 0;
}
